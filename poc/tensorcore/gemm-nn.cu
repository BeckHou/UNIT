#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>

#include "../util.h"

#define N 128
#define M 128
#define K 128

#define KBLOCK 4

using namespace nvcuda;

__global__ void vanilla(half *a, half *b, float *c) {
  int x = blockIdx.y;
  int y = blockIdx.x;

  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float, void> c_frag;
  wmma::fill_fragment(c_frag, 0.0f);

  for (int k = 0; k < K; k += 16) {
    wmma::load_matrix_sync(a_frag, a + (x * 16) * K + k, K);
    wmma::load_matrix_sync(b_frag, b + k * M + y * 16, M);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  }

  wmma::store_matrix_sync(c + (x * 16) * M + (y * 16), c_frag, M, wmma::mem_row_major);
}


__global__ void splitk(half * __restrict__ a, half * __restrict__ b, float * __restrict__ c) {
  int x = blockIdx.y;
  int y = blockIdx.x;
  __shared__ float spad[KBLOCK * 16 * 16];

  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, float, void> c_frag;
  wmma::fill_fragment(c_frag, 0.0f);

  for (int k_inner = 0; k_inner < (K / KBLOCK); k_inner += 16) {
    int k = threadIdx.y * (K / KBLOCK) + k_inner;
    wmma::load_matrix_sync(a_frag, a + (x * 16) * K + k, K);
    wmma::load_matrix_sync(b_frag, b + k * M + y * 16, M);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  }


  wmma::store_matrix_sync(spad + 16 * 16 * threadIdx.y, c_frag, 16, wmma::mem_row_major);

  __syncthreads();

  int workidx = 32 * threadIdx.y + threadIdx.x;
  int workload = (16 * 16) / (32 * KBLOCK);

  for (int i = 0; i < workload; ++i) {
    #pragma UNROLL
    for (int j = 1; j < KBLOCK; ++j) {
      spad[workidx * workload + i] += spad[j * 16 * 16 + workidx * workload + i];
    }
    int xx = (workidx * workload + i) % 16;
    int yy = (workidx * workload + i) / 16;
    c[((x * 16) + xx) * M + (y * 16) + yy] = spad[workidx * workload + i];
  }

}


__global__ void shared_mem(half * __restrict__ a, half * __restrict__ b, float * __restrict__ c) {
  int x = blockIdx.y;
  int y = blockIdx.x;
  __shared__ float spad[KBLOCK * 4 * 16 * 16];

  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag[2][2];
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag[2][2];
  wmma::fragment<wmma::accumulator, 16, 16, 16, float, void> c_frag[2][2];

  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 2; ++j) {
      wmma::fill_fragment(c_frag[i][j], 0.0f);
    }
  }

  for (int k_inner = 0; k_inner < (K / KBLOCK); k_inner += 32) {
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 2; ++j) {
        int k = threadIdx.y * (K / KBLOCK) + k_inner;
        wmma::load_matrix_sync(a_frag[i][j], a + ((x * 32) + (i * 16)) * K + k + j * 16, K);
        wmma::load_matrix_sync(b_frag[i][j], b + (k + i * 16) * M + y* 32 + j * 16, M);
      }
    }
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 2; ++j) {
        for (int k = 0; k < 2; ++k) {
          wmma::mma_sync(c_frag[i][j], a_frag[i][k], b_frag[k][j], c_frag[i][j]);
        }
      }
    }
  }


  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 2; ++j) {
      wmma::store_matrix_sync(spad + 4 * 16 * 16 * threadIdx.y + (i * 2 + j) * 256,
                              c_frag[i][j], 16, wmma::mem_row_major);
    }
  }

  __syncthreads();

  //int workidx = 32 * threadIdx.y + threadIdx.x;
  int workload = (32 * 32) / (32 * KBLOCK);

  for (int i = 0; i < workload; ++i) {
    int idx = threadIdx.y * 256 + (threadIdx.x / 2) * 16 + i + (threadIdx.x % 2) * 8;
    for (int j = 1; j < KBLOCK; ++j) {
      spad[idx] += spad[j * (256 * 4) + idx];
    }
    int xx = idx / 32;
    int yy = idx % 32;
    c[(x * 32 + xx) * M + (y * 32 + yy)] = spad[idx];
  }

}


half a[N * K], b[M * K];
float c[N * M], ref[N * M];

template<typename T>
void print(int n, int m, const T* a) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < m; ++j) {
      if (j) std::cout << " ";
      std::cout << a[i * m + j];
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

template<>
void print(int n, int m, const half* a) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < m; ++j) {
      if (j) std::cout << " ";
      std::cout << __half2float(a[i * m + j]);
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

void compare(int n, float *c, float *ref) {
  for (int i = 0; i < n; ++i) {
    if (fabs(c[i] - ref[i]) / ref[i] > 1e-3) {
      std::cout << i  << "\n" << c[i] << ", expect: " << ref[i] << " " << fabs(c[i] - ref[i]) / ref[i] << std::endl;
      throw;
    }
  }

}

int main() {
  hipDeviceProp_t prop;
  assert(hipSuccess == hipGetDeviceProperties(&prop, 0));
  std::cout << "Warp size is: " <<  prop.warpSize << std::endl;

  for (int i = 0; i < N * K; ++i)
    a[i] = __float2half((float)(rand() % 100) / 100.);
  for (int i = 0; i < K * M; ++i)
    b[i] = __float2half((float)(rand() % 100) / 100.);
  for (int i = 0; i < N; ++i)
    for (int j = 0; j < M; ++j) {
      ref[i * M + j] = 0.0;
      for (int ko = 0; ko < KBLOCK; ++ko) {
        float sub = 0.0;
        for (int ki = 0; ki < K / KBLOCK; ki += 16) {
          float sum = 0;
          for (int kii = 0; kii < 16; ++kii) {
            int k = ko * (K / KBLOCK) + ki + kii;
            sum += __half2float(a[i * K + k]) * __half2float(b[k * M + j]);
          }
          sub += sum;
        }
        ref[i * M + j] += sub;
      }
    }
  half *dev_a, *dev_b;
  hipMalloc(&dev_a, N * K * sizeof(half));
  hipMalloc(&dev_b, M * K * sizeof(half));
  hipMemcpy(dev_a, a, sizeof a, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof b, hipMemcpyHostToDevice);

  std::cout.precision(5);
  {
    memset(c, 0, sizeof(c));
    float *dev_c;
    hipMalloc(&dev_c, N * M * KBLOCK * sizeof(float));
    hipMemcpy(dev_c, c, sizeof c, hipMemcpyHostToDevice);
    dim3 threads(32, 1, 1);
    dim3 blocks(M / 16, N / 16, 1);
    vanilla<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    begin_roi();
    vanilla<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    float elps = end_roi();
    std::cout << "time elps: " << elps << std::endl;
    hipMemcpy(c, dev_c, sizeof c, hipMemcpyDeviceToHost);
    compare(N * M, c, ref);
    hipFree(dev_c);
  }

  {
    memset(c, 0, sizeof(c));
    float *dev_c;
    hipMalloc(&dev_c, N * M * KBLOCK * sizeof(float));
    hipMemcpy(dev_c, c, sizeof c, hipMemcpyHostToDevice);
    dim3 threads(32, KBLOCK, 1);
    dim3 blocks(M / 16, N / 16);
    splitk<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    begin_roi();
    splitk<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    float elps = end_roi();
    std::cout << "time elps: " << elps << std::endl;
    hipMemcpy(c, dev_c, sizeof c, hipMemcpyDeviceToHost);
    compare(N * M, c, ref);
    hipFree(dev_c);
  }

  {
    memset(c, 0, sizeof(c));
    float *dev_c;
    hipMalloc(&dev_c, N * M * KBLOCK * sizeof(float));
    hipMemcpy(dev_c, c, sizeof c, hipMemcpyHostToDevice);
    dim3 threads(32, KBLOCK, 1);
    dim3 blocks(M / 32, N / 32);
    shared_mem<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    begin_roi();
    shared_mem<<<blocks, threads>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();
    float elps = end_roi();
    std::cout << "time elps: " << elps << std::endl;
    std::cout << (N * M * K) / elps / 1000. << std::endl;
    hipMemcpy(c, dev_c, sizeof c, hipMemcpyDeviceToHost);
    compare(N * M, c, ref);
    hipFree(dev_c);
  }


  //print(N, M, a);
  //print(N, M, b);
  //print(N, K, c);
  //print(N, M, ref);
  return 0;
}
