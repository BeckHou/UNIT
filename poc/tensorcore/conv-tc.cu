#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <cassert>
#include "hipDNN.h"

#include "../util.h"

#define checkCudnnErr(x)                     \
  do {                                       \
    auto ret = x;                            \
    if (ret != HIPDNN_STATUS_SUCCESS) {       \
      std::cerr << hipdnnGetErrorString(ret); \
      assert(false);                         \
    }                                        \
  } while (false)

#define checkCudaErr(x)          \
  do {                           \
    auto ret = x;                \
    assert(ret == hipSuccess); \
  } while (false)

int main() {
  hipdnnHandle_t handle;

  checkCudnnErr(hipdnnCreate(&handle));

  hipdnnTensorDescriptor_t xDesc, yDesc;
  hipdnnFilterDescriptor_t wDesc;
  hipdnnConvolutionDescriptor_t convDesc;

  // Create your tensor descriptors:
  checkCudnnErr( hipdnnCreateTensorDescriptor( &xDesc ));
  checkCudnnErr( hipdnnCreateFilterDescriptor( &wDesc ));
  checkCudnnErr( hipdnnCreateTensorDescriptor( &yDesc ));
  checkCudnnErr( hipdnnCreateConvolutionDescriptor( &convDesc ));

  int dimA[4];
  int dimB[4];
  std::cin >> dimA[0] >> dimA[1] >> dimA[2] >> dimA[3];
  std::cin >> dimB[0] >> dimB[1] >> dimB[2] >> dimB[3];

  int pad[2] = {0, 0};
  int stride[2];
  int dilation[2] = {1, 1};

  std::cin >> stride[0] >> stride[1];

  int dimY[4] = {dimA[0], dimB[0], (dimA[2] - dimB[2]) / stride[0] + 1, (dimA[3] - dimB[3]) / stride[1] + 1};
  int dtype_x;
  std::cin >> dtype_x;
  hipdnnDataType_t dtype = (hipdnnDataType_t) dtype_x;

  checkCudnnErr( hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NCHW, dtype,
    dimA[0], dimA[1], dimA[2], dimA[3]) );
  assert(dimA[1] == dimB[1]);
  checkCudnnErr( hipdnnSetFilter4dDescriptor(wDesc, dtype, HIPDNN_TENSOR_NCHW,
    dimB[0], dimB[1], dimB[2], dimB[3]) );
  checkCudnnErr( hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, dtype,
    dimY[0], dimY[1], dimY[2], dimY[3]) );

  void *x = nullptr, *w = nullptr, *y = nullptr;
  // Allocate and initialize tensors (again, only the input tensor is shown):
  checkCudaErr(hipMalloc(&x, dimA[0] * dimA[1] * dimA[2] * dimA[3] * sizeof(float)));
  checkCudaErr(hipMalloc(&w, dimB[0] * dimB[1] * dimB[2] * dimB[3] * sizeof(float)));
  checkCudaErr(hipMalloc(&y, dimY[0] * dimY[1] * dimY[2] * dimY[3] * sizeof(float)));


  // Set the compute data type (below as HIPDNN_DATA_FLOAT):
  checkCudnnErr( hipdnnSetConvolution2dDescriptor(convDesc,
                                                 pad[0], pad[1],
                                                 stride[0], stride[1],
                                                 dilation[0], dilation[1],
                                                 HIPDNN_CROSS_CORRELATION, dtype));
  checkCudnnErr( hipdnnSetConvolutionGroupCount(convDesc, 1) );

  int algo_x, tc;
  std::cin >> algo_x >> tc;

  if (tc) {
    // Set the math type to allow cuDNN to use Tensor Cores:
    checkCudnnErr( hipdnnSetConvolutionMathType(convDesc, HIPDNN_TENSOR_OP_MATH) );
  }

  // Choose a supported algorithm:
  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  algo = (hipdnnConvolutionFwdAlgo_t) algo_x;

  size_t workSpaceSize;
  void *workSpace = nullptr;
  assert(handle);
  assert(xDesc);
  assert(wDesc);
  assert(yDesc);
  assert(convDesc);
  // Allocate your workspace:
  checkCudnnErr( hipdnnGetConvolutionForwardWorkspaceSize(handle, xDesc, 
                                                         wDesc, convDesc,
                                                         yDesc, algo, &workSpaceSize) );

  if (workSpaceSize > 0) {
    hipMalloc(&workSpace, workSpaceSize);
  }

  float alpha = 1.0;
  float beta = 0.0;

  assert(x);
  assert(w);
  assert(y);
  // Invoke the convolution:
  checkCudnnErr( hipdnnConvolutionForward(handle, (void*)(&alpha),
                                         xDesc, x,
                                         wDesc, w, convDesc, algo,
                                         workSpace, workSpaceSize,
                                         (void*)(&beta),
                                         yDesc, y) );
  hipDeviceSynchronize();
  begin_roi();
  for (int i = 0; i < 10; ++i) {
    checkCudnnErr( hipdnnConvolutionForward(handle, (void*)(&alpha),
                                           xDesc, x,
                                           wDesc, w, convDesc, algo,
                                           workSpace, workSpaceSize,
                                           (void*)(&beta),
                                           yDesc, y) );
    checkCudaErr(hipDeviceSynchronize());
  }
  double elps = end_roi();
  elps /= 10.;
  std::cout << "Exec: " << elps << "us" << std::endl;
  std::cout << ((double) dimY[0] * dimY[1] * dimY[2] * dimY[3] * dimB[1] * dimB[2] * dimB[3]) / elps / 1000.
            << " GFLOP/s" << std::endl;

  return 0;
}
