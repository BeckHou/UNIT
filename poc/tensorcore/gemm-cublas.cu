#include "hip/hip_runtime.h"
#include <cassert>
#include <hipblas.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "../util.h"

int m, n, k;

int main() {
  //std::cin >> m >> n >> k;
  m = 128;
  n = 768;
  k = 3072;
  hipblasHandle_t handle;
  half alpha = __float2half(1.0f);
  half *a, *b;
  float *c;
  hipMalloc(&a, m * k * sizeof(half));
  hipMalloc(&b, n * k * sizeof(half));
  hipMalloc(&c, n * m * sizeof(float));
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
  
  assert(hipblasGemmEx(
    handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    m, n, k, &alpha,
    a, HIP_R_16F, m,
    b, HIP_R_16F, k, &alpha,
    c, HIP_R_32F, m, HIP_R_32F, HIPBLAS_GEMM_DEFAULT) == HIPBLAS_STATUS_SUCCESS);
  hipDeviceSynchronize();
  begin_roi();
  assert(hipblasGemmEx(
    handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    m, n, k, &alpha,
    a, HIP_R_16F, m,
    b, HIP_R_16F, k, &alpha,
    c, HIP_R_32F, m, HIP_R_32F, HIPBLAS_GEMM_DEFAULT) == HIPBLAS_STATUS_SUCCESS);
  hipDeviceSynchronize();
  float elps = end_roi();
  std::cout << (n * m * k) / elps / 1000 << std::endl;
  std::cout << elps << std::endl;
  return 0;
}
